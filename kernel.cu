#include<iostream>
#include <hip/hip_runtime.h>


using namespace std;

#define s 96

__global__ void square(int *a, int *b)
{
	int i = threadIdx.x;
	
	if(i<s)
		b[i] = a[i] * a[i] * a[i]; 
	
}


int main()
{
	int *a,*b, i;
	a = (int *)malloc(s * sizeof(int));
	b = (int *)malloc(s * sizeof(int));

	int *d_a, *d_b;
	hipMalloc(&d_a, s * sizeof(int));
	hipMalloc(&d_b, s * sizeof(int));

	for (i = 0; i < s; i++)
	{
		a[i] = i;
	}

	hipMemcpy(d_a, a, s * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, s * sizeof(int), hipMemcpyHostToDevice);

	
	square<<< 1, s >>>(d_a,d_b);

	hipMemcpy(a, d_a, s * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(b, d_b, s * sizeof(int), hipMemcpyDeviceToHost);

	for (i = 0; i < s; i++)
	{
		cout <<i<<":"<< b[i] << ",";
	}

	free(a);
	free(b);

	return 0;
}